#include "hip/hip_runtime.h"
#include "../test_util.hpp"
#include "../cuda/cuda_test_util.cuh"
#include "../cuda/cuda_test_fn.cuh"
#include "ivarp/run_prover.hpp"

namespace {
	using namespace ivarp;
	using namespace ivarp::args;

	const auto G = x0 * sqrt(1_Z-square(x0)) + asin(x0);
    const auto est = G(maximum(x1 - x0 - 1_Z, -1_Z)) - G(ensure_expr(-1_Z)) + 2_Z*square(x0) - x0*x1;

	TEST_CASE("[ivarp][issue#9][cuda] asin test") {
		IDouble widths[] = {
			{0.0, 0.001}, {0.001, 0.01}, {0.01, 0.1}, {0.1, 0.5}, {0.5, 0.99}
		};

		Array<IDouble, 2> r1{IDouble{-1.0, 1.0}, IDouble{-1.0, 1.0}};
		for(int i = 0; i < 20; ++i) {
			cuda_test_fn(asin(x0), 16384, r1, std::begin(widths), std::end(widths));
			cuda_test_fn(G, 16384, r1, std::begin(widths), std::end(widths));
			cuda_test_fn(est, 16384, r1, std::begin(widths), std::end(widths));
		}
	}
}

